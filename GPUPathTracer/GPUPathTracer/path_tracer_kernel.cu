#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "driver_functions.h"
#include "sm_13_double_functions.h"


#include "cutil_math.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <cassert>
#include <ctime>
#include <iostream>
#include <iomanip>
#include <limits>

#include "basic_math.h"

#include "image.h"
#include "sphere.h"
#include "ray.h"
#include "camera.h"

#include "cuda_safe_call.h"

#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>

// Settings:
#define BLOCK_SIZE 256 // Number of threads in a block.
#define MAX_TRACE_DEPTH 5 // TODO: Put settings somewhere else and don't make them defines.

// Numeric constants:
#define PI 3.1415926535897932384626422832795028841971
#define SQRT_OF_ONE_THIRD 0.5773502691896257645091487805019574556476



/*
__device__ float floatInfinity() {
	const unsigned long long ieee754inf = 0x7f800000; // Change the 7 to an f for negative infinity.
	return __longlong_as_float(ieee754inf);
}

__device__ double doubleInfinity() {
	const unsigned long long ieee754inf = 0x7ff0000000000000; // Change the 7 to an f for negative infinity.
	return __longlong_as_double(ieee754inf);
}
*/

__host__ __device__
unsigned int hash(unsigned int a) {
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//E is eye, C is view, U is up
__global__ void raycast_from_camera_kernel(float3 E, float3 C, float3 U, float2 fov, float2 resolution, int numPixels, Ray* rays) {

	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int pixelIndex = BLOCK_SIZE * bx + tx;
	bool validIndex = (pixelIndex < numPixels);

	//get x and y coordinates of pixel
	int y = int(pixelIndex/resolution.y);
	int x = pixelIndex - (y*resolution.y);
	
	if (validIndex) {
		//more compact version, in theory uses fewer registers but horrendously unreadable
		float3 PmE = (E+C) + (((2*(x/(resolution.x-1)))-1)*((cross(C,U)*float(length(C)*tan(fov.x*(PI/180))))/float(length(cross(C,U))))) + (((2*(y/(resolution.y-1)))-1)*((cross(cross(C,U), C)*float(length(C)*tan(-fov.y*(PI/180))))/float(length(cross(cross(C,U), C))))) -E;
		rays[pixelIndex].direction =  normalize(E + (float(200)*(PmE))/float(length(PmE)));

		//more legible version
		/*float CD = length(C);

		float3 A = cross(C,U);
		float3 B = cross(A,C);
		float3 M = E+C;
		float3 H = (A*float(CD*tan(fov.x*(PI/180))))/float(length(A));
		float3 V = (B*float(CD*tan(-fov.y*(PI/180))))/float(length(B));
		
		float sx = x/(resolution.x-1);
		float sy = y/(resolution.y-1);

		float3 P = M + (((2*sx)-1)*H) + (((2*sy)-1)*V);
		float3 PmE = P-E;

		rays[pixelIndex].direction =  normalize(E + (float(200)*(PmE))/float(length(PmE)));
		rays[pixelIndex].origin = E;*/

		//accumulatedColors[pixelIndex] = rays[pixelIndex].direction;	//test code, should output green/yellow/black/red if correct
	}
}

__host__ __device__
//assumes that ray is already transformed into sphere's object space, returns -1 if no intersection
float sphereIntersectionTest(const Ray & ray, const Sphere & sphere, float3 & intersectionPoint, float3 & normal) {

	// http://en.wikipedia.org/wiki/Discriminant
	// http://mathworld.wolfram.com/QuadraticFormula.html
	// http://en.wikipedia.org/wiki/Ray_tracing_%28graphics%29

	normal = make_float3(0,0,0);

	Ray transformedRay;
	transformedRay.origin = ray.origin - sphere.position;
	transformedRay.direction = ray.direction;

	float A = dot(transformedRay.direction, transformedRay.direction);
	float B = 2.0f*dot(transformedRay.direction, transformedRay.origin);
	float C = dot(transformedRay.direction, transformedRay.origin) - (sphere.radius*sphere.radius);

	float discriminant = (B*B)-(4*A*C);
	if(discriminant<0){
		return -1;
	}

	float discriminantSqrt = sqrtf(discriminant);
	float q;
	if(B<0){
        q = (-B - discriminantSqrt) * 0.5; // Changed from / 2.0 to * 0.5 for slightly better performance, although maybe the compiler would do this automatically.
    }else{
        q = (-B + discriminantSqrt) * 0.5;
	}

	
	float t0 = q/A;
    float t1 = C/q;

	// Make t0 the first intersection distance along the ray, and t1 the second:
	if(t0>t1){
		// Swap t0 and t1:
		float temp = t0;
		t0 = t1;
		t1 = temp;
    }

	if(t1<0){
		// Both distances are negative. 
		return -1;
    }

	if(t0<0){
		intersectionPoint = ray.origin + t1*ray.direction;
		normal = normalize(intersectionPoint - sphere.position);
		return t1;
	}else{
		intersectionPoint = ray.origin + t0*ray.direction;
		normal = normalize(intersectionPoint - sphere.position);
		return t0;
	}
}

__host__ __device__
float3 cosineWeightedDirectionInHemisphere(float3 normal, float xi1, float xi2) {

    float up = sqrt(xi1); // cos(theta)
    float over = sqrt(1.0 - up * up); // sin(theta)
    float around = xi2 * PI;

	// Find any two perpendicular directions:
	// Either all of the components of the normal are equal to the square root of one third, or at least one of the components of the normal is less than the square root of 1/3.
	float3 directionNotNormal;
	if (normal.x < SQRT_OF_ONE_THIRD) { 
		directionNotNormal = make_float3(1, 0, 0);
	} else if (normal.y < SQRT_OF_ONE_THIRD) { 
		directionNotNormal = make_float3(0, 1, 0);
	} else {
		directionNotNormal = make_float3(0, 0, 1);
	}
	float3 perpendicular1 = cross(normal, directionNotNormal);
	float3 perpendicular2 = cross(normal, perpendicular1);
  
    return up * normal + std::cos(around) * over * perpendicular1 + std::sin(around) * over * perpendicular2;

}

__global__ void trace_ray_kernel(int numSpheres, Sphere* spheres, int numPixels, Ray* rays, int rayDepth, float3* notAbsorbedColors, float3* accumulatedColors, unsigned long seed) {

//__shared__ float4 something[BLOCK_SIZE]; // 256 (threads per block) * 4 (floats per thread) * 4 (bytes per float) = 4096 (bytes per block)

	// Duplicate code:
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int pixelIndex = BLOCK_SIZE * bx + tx;
	bool validIndex = (pixelIndex < numPixels);

	thrust::default_random_engine rng( hash(seed) * hash(pixelIndex) * hash(rayDepth) );
	thrust::uniform_real_distribution<float> uniformDistribution(0,1);

	if (validIndex) {

		float bestT = 123456789; // floatInfinity(); // std::numeric_limits<float>::infinity();
		float3 bestIntersectionPoint;// = make_float3(0,0,0);
		float3 bestNormal;// = make_float3(0,0,0);
		int bestSphereIndex = -1;
		for (int i = 0; i < numSpheres; i++) {
			float3 intersectionPoint;
			float3 normal;
			float t = sphereIntersectionTest(rays[pixelIndex], spheres[i], intersectionPoint, normal);
			if (t > 0 && t < bestT) {
				bestT = t;
				bestIntersectionPoint = intersectionPoint;
				bestNormal = normal;
				bestSphereIndex = i;
			}
		}
		if (bestSphereIndex >= 0) {
			accumulatedColors[pixelIndex] += notAbsorbedColors[pixelIndex] * spheres[bestSphereIndex].emittedColor;
			notAbsorbedColors[pixelIndex] *= spheres[bestSphereIndex].diffuseColor; // TODO: Use Russian roulette instead of simple multipliers!
			// Choose a new ray direction:
			float randomFloat1 = uniformDistribution(rng); 
			float randomFloat2 = uniformDistribution(rng); 
			float3 newRayDirection = cosineWeightedDirectionInHemisphere(bestNormal, randomFloat1, randomFloat2);
			float rayBiasDistance = 0.0001;
			rays[pixelIndex].origin = bestIntersectionPoint + rayBiasDistance * bestNormal; // TODO: Bias ray in the other direction if the new ray is transmitted.
			rays[pixelIndex].direction = newRayDirection;
		} else {
			float3 backgroundColor = make_float3(0.3, 0.5, 0.8);
			accumulatedColors[pixelIndex] += notAbsorbedColors[pixelIndex] * backgroundColor;
			notAbsorbedColors[pixelIndex] = make_float3(0,0,0); // The ray now has zero weight. TODO: Terminate the ray.
		}

		/*
		// Generate a random number:
		// TODO: Generate more random numbers at a time to speed this up significantly!
		float randomFloat = uniformDistribution(rng); 

		if (randomFloat < 0.5) {
			accumulatedColors[pixelIndex] = rays[pixelIndex].direction;
		} else {
			accumulatedColors[pixelIndex] = make_float3(0,0,0);
		}
		*/

	}

}

extern "C"
void launch_kernel(int numSpheres, Sphere* spheres, int numPixels, Color* pixels, Ray* rays, int counter, Camera* rendercam) {
	
	// Configure grid and block sizes:
	int threadsPerBlock = BLOCK_SIZE;
	// Compute the number of blocks required, performing a ceiling operation to make sure there are enough:
	int blocksPerGrid = (numPixels + threadsPerBlock - 1) / threadsPerBlock;


	// Initialize color arrays:
	Color* tempNotAbsorbedColors = (Color*)malloc(numPixels * sizeof(Color));
	Color* tempAccumulatedColors = (Color*)malloc(numPixels * sizeof(Color));
	for (int i = 0; i < numPixels; i++) {
		tempNotAbsorbedColors[i] = make_float3(1,1,1);
		tempAccumulatedColors[i] = make_float3(0,0,0);
	}
	Color* notAbsorbedColors = NULL;
	Color* accumulatedColors = NULL;
	CUDA_SAFE_CALL( hipMalloc((void**)&notAbsorbedColors, numPixels * sizeof(Color)) );
	CUDA_SAFE_CALL( hipMalloc((void**)&accumulatedColors, numPixels * sizeof(Color)) );
	CUDA_SAFE_CALL( hipMemcpy( notAbsorbedColors, tempNotAbsorbedColors, numPixels * sizeof(Color), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( accumulatedColors, tempAccumulatedColors, numPixels * sizeof(Color), hipMemcpyHostToDevice) );
	free(tempNotAbsorbedColors);
	free(tempAccumulatedColors);


	//only launch raycast from camera kernel if this is the first ever pass!
	// I think we'll have to run this every pass if we want to do anti-aliasing using jittering.
	// Also, if we don't want to re-compute the camera rays, we'll need a separate array for secondary rays.
	//if (counter == 0) {
	raycast_from_camera_kernel<<<blocksPerGrid, threadsPerBlock>>>(rendercam->position, rendercam->view, rendercam->up, rendercam->fov, rendercam->resolution, numPixels, rays);
	//}


	for (int rayDepth = 0; rayDepth < MAX_TRACE_DEPTH; rayDepth++) {
		trace_ray_kernel<<<blocksPerGrid, threadsPerBlock>>>(numSpheres, spheres, numPixels, rays, rayDepth, notAbsorbedColors, accumulatedColors, counter);
	}



	// Copy the accumulated colors from the device into the host image:
	CUDA_SAFE_CALL( hipMemcpy( pixels, accumulatedColors, numPixels * sizeof(Color), hipMemcpyDeviceToHost) );


	// Clean up:
	CUDA_SAFE_CALL( hipFree( notAbsorbedColors ) );
	CUDA_SAFE_CALL( hipFree( accumulatedColors ) );

}
