#include "hip/hip_runtime.h"
#include "cutil_math.h"
#include <math.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "image.h"
#include "sphere.h"
#include "ray.h"

#include "cuda_safe_call.h"


#define BLOCK_SIZE 256 // Number of threads in a block.


__global__ void trace_ray_kernel(int numSpheres, Sphere* spheres, int numPixels, Ray* rays, float3* notAbsorbedColors, float3* accumulatedColors) {

	__shared__ float4 something[BLOCK_SIZE]; // 256 (threads per block) * 4 (floats per thread) * 4 (bytes per float) = 4096 (bytes per block)

	int bx = blockIdx.x;
	int tx = threadIdx.x;

	int somethingIndex = BLOCK_SIZE * bx + tx;
	bool validIndex = (somethingIndex < 999);

}

extern "C"
void launch_kernel(int numSpheres, Sphere* spheres, Image* image, Ray* rays) {
	
	// Configure grid and block sizes and launch the kernel:
	int threadsPerBlock = BLOCK_SIZE;
	// Compute the number of blocks required, performing a ceiling operation to make sure there are enough:
	int blocksPerGrid = (image->numPixels + threadsPerBlock - 1) / threadsPerBlock;

	Color* tempNotAbsorbedColors = (Color*)malloc(image->numPixels * sizeof(Color));
	Color* tempAccumulatedColors = (Color*)malloc(image->numPixels * sizeof(Color));
	Color* notAbsorbedColors = NULL;
	Color* accumulatedColors = NULL;
	hipMalloc((void**)&notAbsorbedColors, image->numPixels * sizeof(Color));
	hipMalloc((void**)&accumulatedColors, image->numPixels * sizeof(Color));
	CUDA_SAFE_CALL( hipMemcpy( notAbsorbedColors, tempNotAbsorbedColors, image->numPixels * sizeof(tempNotAbsorbedColors), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( accumulatedColors, tempAccumulatedColors, image->numPixels * sizeof(tempAccumulatedColors), hipMemcpyHostToDevice) );
	free(tempNotAbsorbedColors);
	free(tempAccumulatedColors);

	trace_ray_kernel<<<blocksPerGrid, threadsPerBlock>>>(numSpheres, spheres, image->numPixels, rays, notAbsorbedColors, accumulatedColors);

}
