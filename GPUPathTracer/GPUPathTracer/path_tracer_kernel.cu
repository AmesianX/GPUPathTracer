#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "driver_functions.h"
#include "sm_13_double_functions.h"


#include "cutil_math.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <cassert>
#include <ctime>
#include <iostream>
#include <iomanip>
#include <limits>

#include "basic_math.h"

#include "image.h"
#include "sphere.h"
#include "ray.h"
#include "camera.h"

#include "cuda_safe_call.h"

#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>

// Settings:
#define BLOCK_SIZE 256 // Number of threads in a block.
#define MAX_TRACE_DEPTH 15 // TODO: Put settings somewhere else and don't make them defines.
#define RAY_BIAS_DISTANCE 0.0002 // TODO: Put with other settings somewhere.

// Numeric constants, copied from BasicMath:
#define PI                    3.1415926535897932384626422832795028841971
#define TWO_PI				  6.2831853071795864769252867665590057683943
#define SQRT_OF_ONE_THIRD     0.5773502691896257645091487805019574556476



/*
__device__ float floatInfinity() {
	const unsigned long long ieee754inf = 0x7f800000; // Change the 7 to an f for negative infinity.
	return __longlong_as_float(ieee754inf);
}

__device__ double doubleInfinity() {
	const unsigned long long ieee754inf = 0x7ff0000000000000; // Change the 7 to an f for negative infinity.
	return __longlong_as_double(ieee754inf);
}
*/

__host__ __device__
unsigned int hash(unsigned int a) {
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//E is eye, C is view, U is up
__global__ void raycast_from_camera_kernel(float3 E, float3 C, float3 U, float2 fov, float2 resolution, int numPixels, Ray* rays) {

	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int pixelIndex = BLOCK_SIZE * bx + tx;
	bool validIndex = (pixelIndex < numPixels);

	//get x and y coordinates of pixel
	int y = int(pixelIndex/resolution.y);
	int x = pixelIndex - (y*resolution.y);
	
	if (validIndex) {
		//more compact version, in theory uses fewer registers but horrendously unreadable
		// Now treating FOV as the full FOV, not half, so I multiplied it by 0.5, although I could be missing something.
		// Another optimization we can make is storing the angles in radians.
		float3 PmE = (E+C) + (((2*(x/(resolution.x-1)))-1)*((cross(C,U)*float(length(C)*tan(fov.x*0.5*(PI/180))))/float(length(cross(C,U))))) + (((2*(y/(resolution.y-1)))-1)*((cross(cross(C,U), C)*float(length(C)*tan(-fov.y*0.5*(PI/180))))/float(length(cross(cross(C,U), C))))) - E;
		rays[pixelIndex].origin = E;
		rays[pixelIndex].direction = normalize(PmE);// normalize(E + (float(200)*(PmE))/float(length(PmE)));

		// I wonder how much slower the more legible version actually is. I would lean towards writing clean code before doing optimizations that destroy readability, but it's seems that's not always the point of GPU programming.
		// Also, we can further improve the more legible version with descriptive variable names.

		//more legible version
		/*float CD = length(C);

		float3 A = cross(C,U);
		float3 B = cross(A,C);
		float3 M = E+C;
		float3 H = (A*float(CD*tan(fov.x*0.5*(PI/180))))/float(length(A)); // Now treating FOV as the full FOV, not half, so I multiplied it by 0.5, although I could be missing something.
		float3 V = (B*float(CD*tan(-fov.y*0.5*(PI/180))))/float(length(B)); // Now treating FOV as the full FOV, not half, so I multiplied it by 0.5, although I could be missing something.
		
		float sx = x/(resolution.x-1);
		float sy = y/(resolution.y-1);

		float3 P = M + (((2*sx)-1)*H) + (((2*sy)-1)*V);
		float3 PmE = P-E;

		rays[pixelIndex].direction = normalize(PmE); // normalize(E + (float(200)*(PmE))/float(length(PmE))); // The E + and the 200 weren't necessary.
		rays[pixelIndex].origin = E;*/

		//accumulatedColors[pixelIndex] = rays[pixelIndex].direction;	//test code, should output green/yellow/black/red if correct
	}
}



__host__ __device__
float findGroundPlaneIntersection(float elevation, const Ray & ray, float3 & intersectionPoint, float3 & normal) {
	// Only finds intersections with the top of the plane.

	if (ray.direction.y < 0) {

		double t = (elevation - ray.origin.y) / ray.direction.y;
	
		intersectionPoint = ray.origin + t * ray.direction;
		normal = make_float3(0, 1, 0);
		
		return t;
	}
	
	return -1; // No intersection.
}



__host__ __device__
//assumes that ray is already transformed into sphere's object space, returns -1 if no intersection
float findSphereIntersection(const Sphere & sphere, const Ray & ray, float3 & intersectionPoint, float3 & normal) {




	// Based on math at http://en.wikipedia.org/wiki/Ray_tracing_%28graphics%29

	float3 v = ray.origin - sphere.position; // Sphere position relative to ray origin.
	float vDotDirection = dot(v, ray.direction);
	float radicand = vDotDirection * vDotDirection - (dot(v, v) - sphere.radius * sphere.radius);
	if (radicand < 0) return -1;
	float squareRoot = sqrt(radicand);
	float firstTerm = -vDotDirection;
	float t1 = firstTerm + squareRoot;
	float t2 = firstTerm - squareRoot;

	float t;

	if (t1 < 0 && t2 < 0) { // (t1 < 0.01 && t2 < 0.01) { // Epsilon shouldn't be necessary here if we have a good global ray bias system.
		return -1;
	} else if (t1 > 0 && t2 > 0) { // (t1 >= 0.01 && t2 >= 0.01) { // Epsilon shouldn't be necessary here if we have a good global ray bias system.
		t = min(t1, t2);
	} else {
		t = max(t1, t2);
	}

	intersectionPoint = ray.origin + t * ray.direction;
	normal = normalize(intersectionPoint - sphere.position);
	return t;

















	/*
	// TEST:
	if (sqrt(ray.direction.x * ray.direction.x + ray.direction.y * ray.direction.y) < 0.1) { //if (ray.direction.z < 0 && ray.direction.y < 0 && ray.direction.z < 0) {
		intersectionPoint = ray.origin + 5.0*ray.direction;
		normal = make_float3(0, 0, -1); //normalize(intersectionPoint - sphere.position);
		return 5.0;
	} else {
		return -1;
	}
	*/














	/*
	// http://en.wikipedia.org/wiki/Discriminant
	// http://mathworld.wolfram.com/QuadraticFormula.html
	// http://en.wikipedia.org/wiki/Ray_tracing_%28graphics%29

	normal = make_float3(0,0,0);

	Ray transformedRay;
	transformedRay.origin = ray.origin - sphere.position;
	transformedRay.direction = ray.direction;

	float A = dot(transformedRay.direction, transformedRay.direction);
	float B = 2.0f*dot(transformedRay.direction, transformedRay.origin);
	float C = dot(transformedRay.direction, transformedRay.origin) - (sphere.radius*sphere.radius);

	float discriminant = (B*B)-(4*A*C);
	if(discriminant<0){
		return -1;
	}

	float discriminantSqrt = sqrtf(discriminant);
	float q;
	if(B<0){
        q = (-B - discriminantSqrt) * 0.5; // Changed from / 2.0 to * 0.5 for slightly better performance, although maybe the compiler would do this automatically.
    }else{
        q = (-B + discriminantSqrt) * 0.5;
	}

	
	float t0 = q/A;
    float t1 = C/q;

	// Make t0 the first intersection distance along the ray, and t1 the second:
	if(t0>t1){
		// Swap t0 and t1:
		float temp = t0;
		t0 = t1;
		t1 = temp;
    }

	if(t1<0){
		// Both distances are negative. 
		return -1;
    }

	if(t0<0){
		intersectionPoint = ray.origin + t1*ray.direction;
		normal = normalize(intersectionPoint - sphere.position);
		return t1;
	}else{
		intersectionPoint = ray.origin + t0*ray.direction;
		normal = normalize(intersectionPoint - sphere.position);
		return t0;
	}
	*/
}

__host__ __device__
float3 cosineWeightedDirectionInHemisphere(const float3 & normal, float xi1, float xi2) {

    float up = sqrt(xi1); // cos(theta)
    float over = sqrt(1.0 - up * up); // sin(theta)
    float around = xi2 * TWO_PI;

	// Find any two perpendicular directions:
	// Either all of the components of the normal are equal to the square root of one third, or at least one of the components of the normal is less than the square root of 1/3.
	float3 directionNotNormal;
	if (abs(normal.x) < SQRT_OF_ONE_THIRD) { 
		directionNotNormal = make_float3(1, 0, 0);
	} else if (abs(normal.y) < SQRT_OF_ONE_THIRD) { 
		directionNotNormal = make_float3(0, 1, 0);
	} else {
		directionNotNormal = make_float3(0, 0, 1);
	}
	float3 perpendicular1 = normalize( cross(normal, directionNotNormal) );
	float3 perpendicular2 =            cross(normal, perpendicular1); // Normalized by default.
  
    return ( up * normal ) + ( cos(around) * over * perpendicular1 ) + ( sin(around) * over * perpendicular2 );

}

__global__ void trace_ray_kernel(int numSpheres, Sphere* spheres, int numPixels, Ray* rays, int rayDepth, float3* notAbsorbedColors, float3* accumulatedColors, unsigned long seed) {

//__shared__ float4 something[BLOCK_SIZE]; // 256 (threads per block) * 4 (floats per thread) * 4 (bytes per float) = 4096 (bytes per block)

	// Duplicate code:
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int pixelIndex = BLOCK_SIZE * bx + tx;
	bool validIndex = (pixelIndex < numPixels);

	thrust::default_random_engine rng( hash(seed) * hash(pixelIndex) * hash(rayDepth) );
	thrust::uniform_real_distribution<float> uniformDistribution(0,1);

	if (validIndex) {

		// TODO: Restructure this block! It's a mess. I want polymorphism!

		// Reusables:
		float t;
		float3 intersectionPoint;
		float3 normal;

		float bestT = 123456789; // floatInfinity(); // std::numeric_limits<float>::infinity();
		float3 bestIntersectionPoint;// = make_float3(0,0,0);
		float3 bestNormal;// = make_float3(0,0,0);
		bool bestIsGroundPlane = false;
		bool bestIsSphere = false;
		int bestSphereIndex = -1;

		// Check for ground plane intersection:
		float hardCodedGroundPlaneElevation = -0.8; // TODO: Put with other settings somewhere.
		t = findGroundPlaneIntersection(hardCodedGroundPlaneElevation, rays[pixelIndex], intersectionPoint, normal); // 123456789; // floatInfinity(); // std::numeric_limits<float>::infinity();
		if (t > 0) { // No "<" conditional only because this is being tested before anythign else.
			bestT = t;
			bestIntersectionPoint = intersectionPoint;
			bestNormal = normal;

			bestIsGroundPlane = true;
			bestIsSphere = false;
		}
		
		// Check for sphere intersection:
		for (int i = 0; i < numSpheres; i++) {
			t = findSphereIntersection(spheres[i], rays[pixelIndex], intersectionPoint, normal);
			if (t > 0 && t < bestT) {
				bestT = t;
				bestIntersectionPoint = intersectionPoint;
				bestNormal = normal;

				bestIsGroundPlane = false;
				bestIsSphere = true;

				bestSphereIndex = i;
			}
		}

		if (bestIsGroundPlane || bestIsSphere) {

			if (bestIsGroundPlane) {
				float3 hardCodedGroundPlaneDiffuseColor = make_float3(0.455, 0.43, 0.39);
				//accumulatedColors[pixelIndex] += NOTHING;
				notAbsorbedColors[pixelIndex] *= hardCodedGroundPlaneDiffuseColor;
			} else if (bestIsSphere) {
				accumulatedColors[pixelIndex] += notAbsorbedColors[pixelIndex] * spheres[bestSphereIndex].emittedColor;
				notAbsorbedColors[pixelIndex] *= spheres[bestSphereIndex].diffuseColor;
			}

			// TODO: Use Russian roulette instead of simple multipliers!
				
			// Choose a new ray direction:
			float randomFloat1 = uniformDistribution(rng); 
			float randomFloat2 = uniformDistribution(rng); 
			float3 newRayDirection = cosineWeightedDirectionInHemisphere(bestNormal, randomFloat1, randomFloat2);
			rays[pixelIndex].origin = bestIntersectionPoint + ( RAY_BIAS_DISTANCE * bestNormal ); // TODO: Bias ray in the other direction if the new ray is transmitted.
			rays[pixelIndex].direction = newRayDirection;

		} else {
			float3 hardCodedBackgroundColor = make_float3(0.15, 0.25, 0.4);
			accumulatedColors[pixelIndex] += notAbsorbedColors[pixelIndex] * hardCodedBackgroundColor;
			notAbsorbedColors[pixelIndex] = make_float3(0,0,0); // The ray now has zero weight. TODO: Terminate the ray.
		}



		/*
		// TEST:
		// Generate a random number:
		// TODO: Generate more random numbers at a time to speed this up significantly!
		float randomFloat = uniformDistribution(rng); 

		if (randomFloat < 0.5) {
			accumulatedColors[pixelIndex] = rays[pixelIndex].direction;
		} else {
			accumulatedColors[pixelIndex] = make_float3(0,0,0);
		}
		*/


	}

}

extern "C"
void launch_kernel(int numSpheres, Sphere* spheres, int numPixels, Color* pixels, Ray* rays, int counter, Camera* rendercam) {
	
	// Configure grid and block sizes:
	int threadsPerBlock = BLOCK_SIZE;
	// Compute the number of blocks required, performing a ceiling operation to make sure there are enough:
	int blocksPerGrid = (numPixels + threadsPerBlock - 1) / threadsPerBlock;


	// Initialize color arrays:
	Color* tempNotAbsorbedColors = (Color*)malloc(numPixels * sizeof(Color));
	Color* tempAccumulatedColors = (Color*)malloc(numPixels * sizeof(Color));
	for (int i = 0; i < numPixels; i++) {
		tempNotAbsorbedColors[i] = make_float3(1,1,1);
		tempAccumulatedColors[i] = make_float3(0,0,0);
	}
	Color* notAbsorbedColors = NULL;
	Color* accumulatedColors = NULL;
	CUDA_SAFE_CALL( hipMalloc((void**)&notAbsorbedColors, numPixels * sizeof(Color)) );
	CUDA_SAFE_CALL( hipMalloc((void**)&accumulatedColors, numPixels * sizeof(Color)) );
	CUDA_SAFE_CALL( hipMemcpy( notAbsorbedColors, tempNotAbsorbedColors, numPixels * sizeof(Color), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( accumulatedColors, tempAccumulatedColors, numPixels * sizeof(Color), hipMemcpyHostToDevice) );
	free(tempNotAbsorbedColors);
	free(tempAccumulatedColors);


	//only launch raycast from camera kernel if this is the first ever pass!
	// I think we'll have to run this every pass if we want to do anti-aliasing using jittering.
	// Also, if we don't want to re-compute the camera rays, we'll need a separate array for secondary rays.
	//if (counter == 0) {
	raycast_from_camera_kernel<<<blocksPerGrid, threadsPerBlock>>>(rendercam->position, rendercam->view, rendercam->up, rendercam->fov, rendercam->resolution, numPixels, rays);
	//}


	for (int rayDepth = 0; rayDepth < MAX_TRACE_DEPTH; rayDepth++) {
		trace_ray_kernel<<<blocksPerGrid, threadsPerBlock>>>(numSpheres, spheres, numPixels, rays, rayDepth, notAbsorbedColors, accumulatedColors, counter);
	}



	// Copy the accumulated colors from the device into the host image:
	CUDA_SAFE_CALL( hipMemcpy( pixels, accumulatedColors, numPixels * sizeof(Color), hipMemcpyDeviceToHost) );


	// Clean up:
	CUDA_SAFE_CALL( hipFree( notAbsorbedColors ) );
	CUDA_SAFE_CALL( hipFree( accumulatedColors ) );

}
