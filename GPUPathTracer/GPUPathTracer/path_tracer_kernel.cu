#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include "cutil_math.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <cassert>
#include <ctime>

#include "image.h"
#include "sphere.h"
#include "ray.h"

#include "cuda_safe_call.h"


#define BLOCK_SIZE 256 // Number of threads in a block.


__global__ void set_up_random_number_generator_kernel(hiprandState* states, unsigned long seed, int numPixels) {
    
	// Duplicate code:
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int pixelIndex = BLOCK_SIZE * bx + tx;
	bool validIndex = (pixelIndex < numPixels);

    hiprand_init(seed, pixelIndex, 0, &states[pixelIndex]);
} 


__global__ void trace_ray_kernel(int numSpheres, Sphere* spheres, int numPixels, Ray* rays, float3* notAbsorbedColors, float3* accumulatedColors, hiprandState* globalCurandStates) {

//__shared__ float4 something[BLOCK_SIZE]; // 256 (threads per block) * 4 (floats per thread) * 4 (bytes per float) = 4096 (bytes per block)

	// Duplicate code:
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int pixelIndex = BLOCK_SIZE * bx + tx;
	bool validIndex = (pixelIndex < numPixels);

	if (validIndex) {

		// Generate a random number:
		// TODO: Generate more random numbers at a time to speed this up significantly!
		hiprandState localCurandState = globalCurandStates[pixelIndex];
		float randomFloat = hiprand_uniform(&localCurandState);
		globalCurandStates[pixelIndex] = localCurandState;

		accumulatedColors[pixelIndex] = make_float3((float)pixelIndex / (float)numPixels, (float)pixelIndex / (float)numPixels / 2.0, (float)pixelIndex / (float)numPixels / 4.0);
		accumulatedColors[pixelIndex] = make_float3(randomFloat, randomFloat, randomFloat);
	}

}

extern "C"
void launch_kernel(int numSpheres, Sphere* spheres, Image* image, Ray* rays) {
	
	// Configure grid and block sizes:
	int threadsPerBlock = BLOCK_SIZE;
	// Compute the number of blocks required, performing a ceiling operation to make sure there are enough:
	int blocksPerGrid = (image->numPixels + threadsPerBlock - 1) / threadsPerBlock;

	// Set up random number generator:
	// TODO: Only do this once, not every frame!
	hiprandState* deviceCurandStates;
    CUDA_SAFE_CALL( hipMalloc((void**)&deviceCurandStates, image->numPixels * sizeof(hiprandState)) );
    set_up_random_number_generator_kernel<<<blocksPerGrid, threadsPerBlock>>>(deviceCurandStates, time(NULL), image->numPixels);

	Color* tempNotAbsorbedColors = (Color*)malloc(image->numPixels * sizeof(Color));
	Color* tempAccumulatedColors = (Color*)malloc(image->numPixels * sizeof(Color));
	Color* notAbsorbedColors = NULL;
	Color* accumulatedColors = NULL;
	CUDA_SAFE_CALL( hipMalloc((void**)&notAbsorbedColors, image->numPixels * sizeof(Color)) );
	CUDA_SAFE_CALL( hipMalloc((void**)&accumulatedColors, image->numPixels * sizeof(Color)) );
	CUDA_SAFE_CALL( hipMemcpy( notAbsorbedColors, tempNotAbsorbedColors, image->numPixels * sizeof(Color), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( accumulatedColors, tempAccumulatedColors, image->numPixels * sizeof(Color), hipMemcpyHostToDevice) );
	free(tempNotAbsorbedColors);
	free(tempAccumulatedColors);

	trace_ray_kernel<<<blocksPerGrid, threadsPerBlock>>>(numSpheres, spheres, image->numPixels, rays, notAbsorbedColors, accumulatedColors, deviceCurandStates);

	// Copy the accumulated colors from the device into the host image:
	CUDA_SAFE_CALL( hipMemcpy( image->pixels, accumulatedColors, image->numPixels * sizeof(Color), hipMemcpyDeviceToHost) );

	CUDA_SAFE_CALL( hipFree( notAbsorbedColors ) );
	CUDA_SAFE_CALL( hipFree( accumulatedColors ) );
	CUDA_SAFE_CALL( hipFree( deviceCurandStates ) );

}
